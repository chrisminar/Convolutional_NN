/***************************************************************************//**
 * \file main.cu
 * \author Christopher Minar (minarc@oregonstate.edu)
 * \brief testing file
 */

#include "operators.h" //contains all device operator tests
#include "back_prop.h" //contains all back propogation kernel tests



int main()
{
	hipDeviceReset();
	test_operators();
	test_back_prop();


	return 0;
}

//todo add a verbose mode to the back_prop indices tests
//todo, run numbers through test kernels

