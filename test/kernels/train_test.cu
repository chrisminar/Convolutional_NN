#include "hip/hip_runtime.h"
/***************************************************************************//**
 * \file train_test.cu
 * \author Christopher Minar (chrisminar@gmail.com)
 * \brief implementation of the training test kernels
 */

#include "train_test.h"

namespace kernels
{

__global__
void calculate_dweight_test(double *dweight, double *input, double *ddot, int *pool_flag,
							int filter_size, int field_height, int field_width,
							int layer_depth, int layer_depth_out, int batch_size,
							int *wi_t, int *oli_t, int *oln_t, int *ili_t,
							int *iln_t, int *ly_t, int *lx_t,
							int *count_t, double *sum_t, int *input_index_t, int *ddot_index_t)
{
	int weight_index = threadIdx.x + blockDim.x * blockIdx.x,						//weight array index
		outer_layer_index = weight_index % (filter_size*filter_size*layer_depth),	//count from 0 to fs*fs*ld for each layer out
		outer_layer_number = weight_index / (filter_size*filter_size*layer_depth),	//outer layer this weight is in
		inner_layer_index = outer_layer_index % (filter_size*filter_size),			//count from 0 to fs*fs for each inner layer
		inner_layer_number = outer_layer_index / (filter_size*filter_size),			//inner layer this weight is in
		layer_y = inner_layer_index/filter_size,									//y position in layer
		layer_x = inner_layer_index%filter_size;									//x position in layer

	wi_t[weight_index] = weight_index;
	oli_t[weight_index] = outer_layer_index;
	oln_t[weight_index] = outer_layer_number;
	ili_t[weight_index] = inner_layer_index;
	iln_t[weight_index] = inner_layer_number;
	ly_t[weight_index] = layer_y;
	lx_t[weight_index] = layer_x;

	if (weight_index >= filter_size*filter_size*layer_depth*layer_depth_out)
		return;

	double sum=0;
	int count=0;
	int input_index;
	int ddot_index;
	int flag=1;
	int loop_index = 0;

	//step through images
	for (int m=0; m<batch_size; m++)
	{
		//step through field y
		for (int j=0; j<field_height; j++)
		{
			//step through field x
			for (int i=0; i<field_width; i++)
			{
				input_index =	m * field_width*field_height*layer_depth +
								inner_layer_number * field_width*field_height +
								j * field_width +
								i;
				ddot_index = 	m * field_width*field_height*layer_depth_out +
								outer_layer_number * field_width*field_height +
								j * field_width +
								i;
				loop_index = m * field_height*field_width +
								j*field_width+
								i;
				input_index_t[weight_index * batch_size*field_height*field_width + loop_index] = input_index;
				ddot_index_t[weight_index * batch_size*field_height*field_width + loop_index] = ddot_index;
				//if were out of bounds, mult this weight*temp by zero
				if ( (layer_y == 0 and j==field_height-1) || (layer_y==filter_size-1 and j==0) || (layer_x == 0 and i==field_width-1) || (layer_x==filter_size-1 and i == 0) ) //todo might be able to cut out this if statement by modifiying the above for loops
					flag = 0;
				else
					flag = 1;
				sum += input[input_index] * ddot[ddot_index] * flag;// * pool_flag[i];
				count +=1*flag;
			}
		}
	}
	dweight[weight_index] = sum;
	sum_t[weight_index] = sum;
	count_t[weight_index] = count;
}

//can't handle things that are not 1x1xsomething
__global__
void calculate_fc_dweight_test(double *dweight, double *input, double *ddot, int filter_size, int field_height, int field_width, int layer_depth, int layer_depth_out, int batch_size)
{
	int weight_index = threadIdx.x + blockDim.x * blockIdx.x,						//weight array index
		outer_layer_index = weight_index % (filter_size*filter_size*layer_depth),	//count from 0 to fs*fs*ld for each layer out
		outer_layer_number = weight_index / (filter_size*filter_size*layer_depth),	//outer layer this weight is in
		inner_layer_index = outer_layer_index % (filter_size*filter_size),			//count from 0 to fs*fs for each inner layer
		inner_layer_number = outer_layer_index / (filter_size*filter_size),			//inner layer this weight is in
		layer_y = inner_layer_index/filter_size,									//y position in layer
		layer_x = inner_layer_index%filter_size;									//x position in layer

	if (weight_index >= filter_size*filter_size*layer_depth*layer_depth_out)
		return;

	double sum=0;
	int input_index;
	int ddot_index;

	//step through images
	for (int m=0; m<batch_size; m++)
	{
		ddot_index =	1*1*layer_depth_out*m + //1x1 should be be field width out and field height out
						1*1*outer_layer_number;
		input_index =	m * field_width*field_height*layer_depth +
						inner_layer_number * field_width*field_height +
						layer_y * field_width +
						layer_x;
		sum += input[input_index] * ddot[ddot_index];
	}
	dweight[weight_index] = sum;
}

__global__
void propogate_ddot_fc_test(double *ddot, double *ddot_upstream, double *weights, double *bias,
						int field_height, int field_width, int layer_depth_out, int filter_size,
						int field_height_us, int field_width_us, int layer_depth_out_us, int batch_size)
{
	//some useful numbers
	int num_pixels_per_ddot_layer = 1,
		num_pixels_per_ddot_image = num_pixels_per_ddot_layer*layer_depth_out,
		num_pixels_per_layer_us = field_width_us*field_height_us,
		num_pixels_per_image_us = num_pixels_per_layer_us*layer_depth_out_us;

	//figure out where this kernel is
	int output_index = threadIdx.x + blockDim.x * blockIdx.x,		// ddot index
		image_number = output_index / (num_pixels_per_image_us),	// image this ddot is in
		image_index = output_index % (num_pixels_per_image_us),		// count from 0 to num_pixels_per_image in this image
		layer_us_number = image_index / (num_pixels_per_layer_us),	// output layer this image is in
		layer_us_index = image_index % (num_pixels_per_layer_us),	// count from 0 to num_pixels_per_layer in this output layer
		field_x = layer_us_index % field_width_us,					// ddot x position in output layer
		field_y = layer_us_index / field_width_us;					// ddot y position in output layer

	//if were outside of the minibatch range return
	if (output_index >= num_pixels_per_image_us * batch_size)
		return;

	//setup dot product
	double sum = 0;
	int ddot_index = 0;
	int weight_index = 0;

	//loop over layer depth
	for (int k=0; k<layer_depth_out; k++)
	{
		ddot_index =	image_number*num_pixels_per_ddot_image +		// past images
						k*num_pixels_per_ddot_layer;					// layers
		weight_index = 	k*field_width*field_height*layer_depth_out +	// layer outs
						layer_us_number*field_height*field_width +		// layer in
						field_y*(field_height/field_height_us)*field_width +	// filter rows
						field_x*(field_width/field_width_us);			// filter columns note: each filter is rotated by 180, which is why we go the the max of the filter layer then subtract off
		sum += ddot[ddot_index] * weights[weight_index];
	}//endk
	ddot_upstream[output_index] = sum; //todo deal with bias
}

//note: not setup for alternate filter sizes, zero padding or strides
//note: not sure if we need a pool flag for this kernel
__global__
void propogate_ddot_conv_test(double *ddot, double *ddot_upstream, double *weights, double *bias,
								int field_height, int field_width, int layer_depth_out, int filter_size,
								int field_height_us, int field_width_us, int layer_depth_out_us, int batch_size) //layer_depth_out_us is the same as layer_depth for the current layer
{
	//some useful numbers
	int num_pixels_per_layer = field_width*field_height,
		num_pixels_per_image = num_pixels_per_layer*layer_depth_out,
		num_pixels_per_layer_us = field_width_us*field_height_us,
		num_pixels_per_image_us = num_pixels_per_layer_us*layer_depth_out_us;

	//figure out where this kernel is
	int output_index = threadIdx.x + blockDim.x * blockIdx.x,		// ddot index
		image_number = output_index / (num_pixels_per_image_us),	// image this ddot is in
		image_index = output_index % (num_pixels_per_image_us),		// count from 0 to num_pixels_per_image in this image
		layer_us_number = image_index / (num_pixels_per_layer_us),	// output layer this image is in
		layer_us_index = image_index % (num_pixels_per_layer_us),	// count from 0 to num_pixels_per_layer in this output layer
		field_x = layer_us_index % field_width_us,					// ddot x position in output layer
		field_y = layer_us_index / field_width_us;					// ddot y position in output layer


	//if were outside of the minibatch range return
	if (output_index >= num_pixels_per_image_us * batch_size)
		return;

	//setup dot product
	double sum = 0;
	int center_pixel_index = 0;
	int weight_index = 0;
	//the weight is going to be filter_size * filter_size * layer depth_in. There is one weight for each output layer
	//loop over output depth
	int filter_half = filter_size/2;
	//loop over layer depth
	for (int k=0; k<layer_depth_out; k++)
	{
		center_pixel_index =	image_number*num_pixels_per_image + 					// past images
								k*num_pixels_per_layer +								// layers
								field_y*(field_height/field_height_us)*field_width +	// rows
								field_x*(field_width/field_height_us);					// columns
		//loop over filter_x
		for (int i=-filter_size/2; i<filter_half; i++)
		{
			//loop over filter_y
			for (int j=-filter_size/2; j<filter_half; j++)
			{
				//check if we are at a boundary to account for zero padding
				//      left                           right                           bottom                     top
				if ( (field_x + i < 0) || (field_x + i >= field_width-1) || (field_y + j < 0) || (field_y + j >= field_height-1) )
				{}
				else
				{
					weight_index = 			k*filter_size*filter_size*layer_depth_out +			// layer outs
											layer_us_number*filter_size*filter_size +		// layer in
											filter_size*filter_size - 1 -					// maximum filter layer
											filter_size*(j+filter_half) -					// filter rows
											(i+filter_half);								// filter columns note: each filter is rotated by 180, which is why we go the the max of the filter layer then subtract off
					sum += ddot[center_pixel_index + j*field_width + i] * weights[weight_index];
				}//endif
			}//endj
		}//endi
	}//endk
	ddot_upstream[output_index] = sum; //todo deal with bias
}

}
