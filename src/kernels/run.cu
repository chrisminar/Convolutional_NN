#include "hip/hip_runtime.h"
/***************************************************************************//**
 * \file run.cu
 * \author Christopher Minar (chrisminar@gmail.com)
 * \brief Declaration of the class layer
 */

#include "run.h"

namespace kernels
{
__global__//todo I'm suspicious of this kernels indexing, looping over layer depth and layer depth out seems wrong
void convolute(double *input, double *temp, double *weights, double *bias, int field_width, int field_height,
		int stride_x, int stride_y, int zero_pad_x, int zero_pad_y, int filter_size, int batch_size, int layer_depth, int layer_depth_out)
{
	//todo currently some parts assume the x-y symmetry
	//todo currently handles zero-pad oddly

	//figure out where this kernel is
	int number_of_filters_per_width = ((field_width - filter_size + 2*zero_pad_x)/stride_x + 1),			//number of filters in each row of a lyer
		number_of_filters_per_input = number_of_filters_per_width*number_of_filters_per_width,				//number of filters in one layer
		total_filter_number = threadIdx.x + blockDim.x * blockIdx.x,										//filter number (no reset)
		image_number = total_filter_number/(number_of_filters_per_input*layer_depth),						//image number were on
		filter_number = total_filter_number%(number_of_filters_per_input),									//filter number in the layer
		filter_x = filter_number % field_width,																//filter number of column in layer
		filter_y = filter_number / field_width;																//filter number of row in layer
	//if were outside of the minibatch range return
	if (total_filter_number >= number_of_filters_per_input * layer_depth * batch_size)
		return;

	//setup dot product
	//this if statement causes crazy divergence
	double sum = 0;
	int center_pixel_index = 0;
	int weight_index = 0;
	//the weight is going to be filter_size x filter_size x layer depth_in. There is one weight for each output layer
	//loop over output depth
	int filter_half = filter_size/2;
	for (int m=0; m<layer_depth_out; m++)
	{
		//loop over input depth
		for (int k=0; k<layer_depth; k++)
		{
			//loop over filter_x
			for (int i=-filter_size/2; i<filter_half; i++)
			{
				//loop over filter_y
				for (int j=-filter_size/2; j<filter_half; j++)
				{
					//check if we are at a boundary
					if (filter_x + i < 0)
					{
						//we're off the left side of the image, do nothing
					}
					else if (filter_x + i >= field_width-1)
					{
						//we're off the right side of the image, this doesn't really account for stride
					}
					else if (filter_y + j < 0)
					{
						//off the bottom
					}
					else if (filter_y + j >= field_height-1)
					{
						//off the top
					}
					else
					{
						//otherwise we have no zero-padding required, dot product away
						//	pixels from...                      past images							        past layers              past rows         past columns
						center_pixel_index = image_number*field_width*field_height*layer_depth +  field_width*field_height*k + field_width*filter_y + filter_x; //note this only works if stride is one with approp zero-padding
						weight_index = filter_size*filter_size*layer_depth*m +  k*filter_size*filter_size + filter_size*(j+filter_half) + (i+filter_half);
						sum += input[center_pixel_index + j*field_width + i] * weights[weight_index];
					}//endif
				}//endj
			}//endi
		}//endk
		temp[image_number*field_width*field_height*layer_depth_out + field_width*field_height*m + field_width * filter_y + filter_x] = sum + bias[m];
		sum = 0;
	}//endm*/
}

__global__
void convolute_FC(double *input, double *temp, double *weights, double *bias, int field_width, int field_height,
				int batch_size, int layer_depth, int layer_depth_out)
{
	//todo currently some parts assume the x-y symmetry

	//useful numbers
	int num_pixel_per_layer = field_width*field_height,
		num_pixel_per_image = num_pixel_per_layer*layer_depth;

	int neuron_index = threadIdx.x + blockDim.x * blockIdx.x,
		image_index = neuron_index / layer_depth_out,
		neuron = neuron_index%layer_depth_out;

	//if were outside of the minibatch range return
	if (neuron_index >= layer_depth_out * batch_size)
		return;

	//setup dot product
	//this if statement causes crazy divergence
	double sum = 0;
	int weight_index = 0;
	int pixel_index = 0;
	//loop over input depth
	for (int k=0; k<layer_depth; k++)
	{
		//loop over field_x
		for (int i=0; i<field_width; i++)
		{
			//loop over filter_y
			for (int j=0; j<field_height; j++)
			{
				weight_index = num_pixel_per_image*neuron +
								num_pixel_per_layer*k +
								field_width*j +
								i;
				pixel_index = num_pixel_per_image*image_index +
								num_pixel_per_layer*k +
								field_width * j +
								i;
				sum += input[pixel_index] * weights[weight_index];
			}//endj
		}//endi
	}//endk
	temp[neuron_index] = sum + bias[neuron];
}

__global__
void sigmoid_activation(double *temp, int field_width, int field_height, int layer_depth_out, int batch_size)
{
	int number_of_pixels_per_image = field_width*field_height*layer_depth_out;
	if (threadIdx.x + blockDim.x * blockIdx.x >= number_of_pixels_per_image * batch_size)
		return;
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	temp[i] =  1 / (1 + exp(-temp[i]));
}

__global__
void pool_input(double *temp, double *output, int field_width, int field_height, int layer_depth_out, int batch_size)
{
	int number_of_pixels_per_image = field_width*field_height*layer_depth_out;
	if (threadIdx.x + blockDim.x * blockIdx.x >= number_of_pixels_per_image * batch_size)
		return;
	int		pool_index = threadIdx.x + blockDim.x * blockIdx.x,						//index for the pool output
			image_number = pool_index/number_of_pixels_per_image,					//what image are we at
			layer_number = (pool_index%number_of_pixels_per_image)/(field_width*field_height),	//what layer are we at
			layer_index = (pool_index%number_of_pixels_per_image)%(field_width*field_height),		//layer index from 0 to xx within each layer
			pool_x = layer_index % (field_width),									//the column we are on
			pool_y = layer_index / (field_width),									//the row we are on
			temp_index = image_number*field_width*field_height*layer_depth_out*4 +	//pixels from previous images...
							layer_number*field_width*field_height*4 +					//from previous layers
							pool_y*field_width*4 +									//rows
							pool_x*2;												//columns
	//perform pooling opertaion
	double max = temp[temp_index];
	if (temp[temp_index+1] > max)
		max = temp[temp_index+1];
	if (temp[temp_index + field_width*2] > max)
		max = temp[temp_index + field_width*2];
	if (temp[temp_index + field_width*2 + 1] > max)
		max = temp[temp_index + field_width*2 + 1];
	output[pool_index] = max;
}
}
