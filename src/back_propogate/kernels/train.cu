#include "hip/hip_runtime.h"
/***************************************************************************//**
 * \file train.cu
 * \author Christopher Minar (chrisminar@gmail.com)
 * \brief Declaration of the training kernels
 */

#include "train.h"

namespace kernels
{

__global__
void calculate_dweight(double *dweight, double *input, double *ddot, int *pool_flag, int filter_size, int field_height, int field_width, int layer_depth, int layer_depth_out, int batch_size)
{
	int weight_index = threadIdx.x + blockDim.x * blockIdx.x,						//weight array index
		outer_layer_index = weight_index % (filter_size*filter_size*layer_depth),	//count from 0 to fs*fs*ld for each layer out
		outer_layer_number = weight_index / (filter_size*filter_size*layer_depth),	//outer layer this weight is in
		inner_layer_index = outer_layer_index % (filter_size*filter_size),			//count from 0 to fs*fs for each inner layer
		inner_layer_number = outer_layer_index / (filter_size*filter_size),			//inner layer this weight is in
		layer_y = inner_layer_index/filter_size,											//y position in layer
		layer_x = inner_layer_index%filter_size;											//x position in layer

	if (weight_index >= filter_size*filter_size*layer_depth*layer_depth_out);
		return;

	double sum=0;
	int input_index;
	int ddot_index;
	int flag=1;

	//step through images
	for (int m=0; m<batch_size; m++)
	{
		//step through field y
		for (int j=0; j<field_height; j++)
		{
			//step through field x
			for (int i=0; i<field_width; i++)
			{
				input_index =	m * field_width*field_height*layer_depth +
								inner_layer_num * field_width*field_height +
								j * field_width +
								i;
				ddot_index = 	m * field_width*field_height*layer_depth_out +
								outer_layer_num * field_width*field_height +
								j * field_width +
								i;

				//if were out of bounds, mult this weight*temp by zero
				if ( (layer_y == 0 and j==field_height-1) || (layer_y==filter_size-1 and j==0) || (layer_x == 0 and i==field_width-1) || (layer_x==filter_size-1 and i == 0) ) //todo might be able to cut out this if statement by modifiying the above for loops
					flag = 0;
				else
					flag = 1;
				sum += input[input_index] * ddot[ddot_index] * flag * pool_flag[i];
			}
		}
	}
	dweight[weight_index] = sum;
}

//can't handle things that are not 1x1xsomething
__global__
void calculate_fc_dweight(double *dweight, double *input, double *dinput, int filter_size, int field_height, int field_width, int layer_depth, int layer_depth_out, int batch_size)
{
	int weights_index = threadIdx.x + blockDim.x * blockIdx.x,
		layer_do_index = weights_index/(field_height*field_width*layer_depth);

	if (weights >= filter_size*filter_size*layer_depth);
		return;

	double sum=0;
	int input_index;
	int ddot_index;

	//step through images
	for (int m=0; m<batch_size; m++)
	{
		ddot_index =	1*1*layer_depth_out*m + //1x1 could be field width out and field height out
						1*1*layer_do_index;
		//step through layer_depth
		for (int k=0; k<layer_depth; k++)
		{
			//step through field y
			for (int j=0; j<field_height; j++)
			{
				//step through field x
				for (int i=0; i<field_width; i++)
				{
					input_index =	m * field_width*field_height*layer_depth +
									k * field_width*field_height +
									j * field_width +
									i;
					sum += input[input_index] * ddot[ddot_index];
				}
			}
		}
	}
	dweight[neuron_index] = sum;
}
}
