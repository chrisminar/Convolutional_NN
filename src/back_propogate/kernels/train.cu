#include "hip/hip_runtime.h"
/***************************************************************************//**
 * \file train.cu
 * \author Christopher Minar (chrisminar@gmail.com)
 * \brief Declaration of the training kernels
 */

#include "train.h"

namespace kernels
{

__global__
void calculate_dweight(double *dweight, double *input, double *ddot, int filter_size, int field_height, int field_width, int layer_depth, int layer_depth_out, int batch_size)
{
	int weight_index = threadIdx.x + blockDim.x * blockIdx.x,						//weight array index
		weight_li_index = weight_index/(filter_size*filter_size*layer_depth),		//counts from 0 to fs*fs*ld for each layer out
		layer_index = weight_li_index/(filter_size*filter_size),					//index of the layer we are on
		layer_y = layer_index/filter_size,											//y position in layer
		layer_x = layer_index%filter_size;											//x position in layer

	if (weight_index >= filter_size*filter_size*layer_depth*layer_depth_out);
		return;

	double sum=0;
	int temp_index;
	double flag=1;

	//step through images
	for (int m=0; m<batch_size; m++)
	{
		//step through layer_depth
		for (int k=0; k<layer_depth; k++)
		{
			//step through field y
			for (int j=0; j<field_height; j++)
			{
				//step through field x
				for (int i=0; i<field_width; i++)
				{
					input_index =	m * field_width*field_height*layer_depth +
									k * field_width*field_height +
									j * field_width +
									i;
					//if were out of bounds, mult this weight*temp by zero
					if ( (layer_y == 0 and j==field_height-1) || (layer_y==filter_size-1 and j==0) || (layer_x == 0 and i==field_width-1) || (layer_x==filter_size-1 and i == 0) ) //todo might be able to cut out this if statement by modifiying the above for loops
						flag = 0;
					else
						flag = 1;
					sum += input[input_index] * ddot[input_index] * flag; //todo ddot should not be input_index or sized the same as input for conv layers
				}
			}
		}
	}
	dweight[weight_index] = sum;
}

//can't handle things that are not 1x1xsomething
__global__
void calculate_fc_dweight(double *dweight, double *input, double *dinput, int filter_size, int field_height, int field_width, int layer_depth, int layer_depth_out, int batch_size)
{
	int weights_index = threadIdx.x + blockDim.x * blockIdx.x,
		layer_do_index = weights_index/(field_height*field_width*layer_depth);

	if (weights >= filter_size*filter_size*layer_depth);
		return;

	double sum=0;
	int input_index;
	int ddot_index;

	//step through images
	for (int m=0; m<batch_size; m++)
	{
		ddot_index =	1*1*layer_depth_out*m + //1x1 could be field width out and field height out
						1*1*layer_do_index;
		//step through layer_depth
		for (int k=0; k<layer_depth; k++)
		{
			//step through field y
			for (int j=0; j<field_height; j++)
			{
				//step through field x
				for (int i=0; i<field_width; i++)
				{
					input_index =	m * field_width*field_height*layer_depth +
									k * field_width*field_height +
									j * field_width +
									i;
					sum += input[input_index] * ddot[ddot_index];
				}
			}
		}
	}
	dweight[neuron_index] = sum;
}
}
