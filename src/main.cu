/***************************************************************************//**
 * \file main.cu
 * \author Christopher Minar (minarc@oregonstate.edu)
 * \brief Main source-file of \c convNet
 */

#include "io.h"
#include "types.h"
#include "image_DB.h"
#include "network.h"

int main()
{
	hipDeviceReset();
	//io::print_gpu_data();
	image_DB idb;
	network ntwrk(&idb);
    io::read_CIFAR10(idb);
	std::string fname = "/scratch/src/convNet/convNet/validation/CIFAR10.yaml";
	io::parse_network_file(fname, ntwrk);
	//ntwrk.print_network_info();
	ntwrk.initialise_layers();
	io::printDeviceMemoryUsage();
	ntwrk.run();
	ntwrk.train_epoch();

    return 0;
}

//todo make print info dump to a file
//todo backprop
//todo is bias accounted for in the fully connected layer?

//todo revisit the dtemp = dsig(Temp) kernels, then update network accordingly
//todo calculate dweight is done for convolutional layers
//todo dweight for pooling
//todo dweight for fully_connected
//todo dweight biases

//general questions:
//should weights be positive?
//what is the best way to introduce bias into our solution?
//why do we have to re cast arrays in network? --> why are the pointers changing between when they are first cast and run?
